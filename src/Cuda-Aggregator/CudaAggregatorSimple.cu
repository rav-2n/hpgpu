#include "hip/hip_runtime.h"
#include "./cuda_helper.cuh"
#include "./DataGeneratorWithCuda.cu"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <iostream>
#include <stdexcept>
#include <limits>

template <typename T, int TBlocksize>
__global__ void aggregate_addition_with_shared_memory(T *, T *, int);

class CudaAggregatorSimple
{

public:
    int addition_with_shared_memory(uint64_t size, uint64_t init)
    {
        int device = 0;

        try
        {
            aggregate_addition<uint64_t, 128>(size, device, init);
        }
        catch (std::runtime_error &error)
        {
            std::cerr << error.what() << "\n";
            CHECK_CUDA(hipDeviceReset());
            return 1;
        }
        CHECK_CUDA(hipDeviceReset());
        return 0;
    }

private:
    template <typename T, int TBlocksize>
    void aggregate_addition(uint64_t n, int dev, T init = 0)
    {

        CHECK_CUDA(hipSetDevice(dev));
        hipDeviceProp_t prop;
        CHECK_CUDA(hipGetDeviceProperties(&prop, dev));
        hipEvent_t cstart, cend;
        CHECK_CUDA(hipEventCreate(&cstart));
        CHECK_CUDA(hipEventCreate(&cend));

        std::cout << getCUDADeviceInformations(dev).str()
                  << "\n\n";

        int numSMs;
        CHECK_CUDA(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, dev));
        dim3 blocks(16 * numSMs); // 16*128 = 2048 (=max resident threads on SM), rule of thumb
        if (blocks.x > ((n - 1) / TBlocksize + 1))
            blocks.x = (n - 1) / TBlocksize + 1;

        // T *h_x = new T[n];
        T h_x[n];
        T *x = nullptr;
        T *y = nullptr;
        T result_gpu = 0;

        CHECK_CUDA(hipMalloc(&x, n * sizeof(T)));
        CHECK_CUDA(hipMalloc(&y, blocks.x * sizeof(T)));

        // init host memory - TODO: USE DATA GENERATOR AFTER INTRODUCING CMAKE
        if (init != 0)
        {
            for (int i = 0; i < n; i++)
                h_x[i] = init;
        }

        else
        {
            int const NBlock = 26;
            int const NThread = 1024;
            DataGeneratorWithCuda::gen<NBlock, NThread>(n, h_x);
        }

        CHECK_CUDA(hipMemcpy(x, h_x, n * sizeof(T), hipMemcpyHostToDevice));

        float milliseconds = 0;
        float min_ms = std::numeric_limits<float>::max();

        CHECK_CUDA(hipMemset(y, 0, sizeof(T)));
        CHECK_CUDA(hipEventRecord(cstart));

        aggregate_addition_with_shared_memory<uint64_t, TBlocksize><<<blocks, TBlocksize>>>(x, y, n);
        aggregate_addition_with_shared_memory<uint64_t, TBlocksize><<<1, TBlocksize>>>(y, y, blocks.x);

        CHECK_CUDA(hipEventRecord(cend));
        CHECK_CUDA(hipEventSynchronize(cend));
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipEventElapsedTime(&milliseconds, cstart, cend));
        if (milliseconds < min_ms)
            min_ms = milliseconds;

        CHECK_CUDA(hipMemcpy(&result_gpu, y, sizeof(T), hipMemcpyDeviceToHost));

        std::cout << "Result (n = " << n << "):\n"
                  << "GPU: " << result_gpu << " (min kernels time = " << min_ms << " ms)\n";

        if (init != 0)
        {
            cout << "expected: " << init * n << "\n"
                 << (init * n != result_gpu ? "MISMATCH!!" : "Success") << "\n";
        }

        cout << "max bandwidth: " << n * sizeof(T) / min_ms * 1e-6 << " GB/s"
        << std::endl;

        // delete[] h_x;
        CHECK_CUDA(hipFree(x));
        CHECK_CUDA(hipFree(y));
        CHECK_CUDA(hipEventDestroy(cstart));
        CHECK_CUDA(hipEventDestroy(cend));
    }
};

template <typename T, int TBlocksize>
__global__ void aggregate_addition_with_shared_memory(T *x, T *y, int n)
{
    __shared__ T sdata[TBlocksize];

    int tid = threadIdx.x;
    int i = blockIdx.x * TBlocksize + threadIdx.x;

    // safeguard
    if (i > n)
        return;

    // store thread local sum in register, initialize with "current value"
    T tsum = x[i];

    // offset for each thread
    int gridsize = gridDim.x * TBlocksize;

    i += gridsize;

    // grid reduce
    while (i < n)
    {
        tsum += x[i];
        i += gridsize;
    }

    sdata[tid] = tsum;

    __syncthreads();

#pragma unroll
    for (unsigned int bs = TBlocksize,
                      bsup = (TBlocksize + 1) / 2; // ceil(TBlocksize/2.0)
         bs > 1;
         bs = bs / 2,
                      bsup = (bs + 1) / 2) // ceil(bs/2.0)
    {
        bool cond = threadIdx.x < bsup                                     // only first half of block is working
                    && (threadIdx.x + bsup) < TBlocksize                   // index for second half must be in bounds
                    && (blockIdx.x * TBlocksize + threadIdx.x + bsup) < n; // if elem in second half has been initialized before
        if (cond)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + bsup];
        }
        __syncthreads();
    }

    __syncthreads();

    if (tid == 0)
        y[blockIdx.x] = sdata[0];
}

int main()
{
    CudaAggregatorSimple cudaAggregatorSimple;
    cudaAggregatorSimple.addition_with_shared_memory(1e8, 1);
    return 0;
}